#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}


__device__ bool computeCov3D(const glm::vec3 &p_world, const glm::vec4 &quat, const glm::vec2 &scale, const float *viewmat, const float4 &intrins, float tan_fovx, float tan_fovy, float* cov3D, float3 &normal) {
	// camera information 
	const glm::mat3 W = glm::mat3(
		viewmat[0],viewmat[1],viewmat[2],
		viewmat[4],viewmat[5],viewmat[6],
		viewmat[8],viewmat[9],viewmat[10]
	); // viewmat 


	const glm::vec3 cam_pos = glm::vec3(viewmat[12], viewmat[13], viewmat[14]); // camera center
	const glm::mat4 P = glm::mat4(
		intrins.x, 0.0, 0.0, 0.0,
		0.0, intrins.y, 0.0, 0.0,
		intrins.z, intrins.w, 1.0, 1.0,
		0.0, 0.0, 0.0, 0.0
	);

	glm::vec3 p_view = W * p_world + cam_pos;
	glm::mat3 R = quat_to_rotmat(quat) * scale_to_mat({scale.x, scale.y, 1.0f}, 1.0f);

#if VIEW_FRUSTUM_CULLING
#if PLUS_R
	const float r = max(glm::length(R[0]), glm::length(R[1])) / p_view.z;
#else
	const float r = 0.0f;
#endif
	const float limx = CLIP_THRESH * tan_fovx + r;
	const float limy = CLIP_THRESH * tan_fovy + r;
	// culing spalt that outside the view frustum
	const float pxpz = (p_view.x) / p_view.z;
	const float pypz = (p_view.y) / p_view.z;
#if HARD_CULLING
	if (pxpz < -limx || pxpz > limx || pypz < -limy || pypz > limy) {
		return false;
	}
#else
	p_view.x = min(limx, max(-limx, pxpz)) * p_view.z;
	p_view.y = min(limy, max(-limy, pypz)) * p_view.z;
#endif
#endif

	glm::mat3 M = glm::mat3(W * R[0], W * R[1], p_view);
	// don't draw if the matrix is singular
	// if (glm::determinant(M) == 0.0f) return false;
	// back face culling ? or parallel face culling?
	glm::vec3 tn = W*R[2];
	float cos = glm::dot(-tn, p_view);
	if (cos == 0.0f) return false;

#if RENDER_AXUTILITY and DUAL_VISIABLE
	float multiplier = cos > 0 ? 1 : -1;
	tn *= multiplier;
#endif

	glm::mat4x3 T = glm::transpose(P * glm::mat3x4(
		glm::vec4(M[0], 0.0),
		glm::vec4(M[1], 0.0),
		glm::vec4(M[2], 1.0)
	));

	cov3D[0] = T[0].x;
	cov3D[1] = T[0].y;
	cov3D[2] = T[0].z;
	cov3D[3] = T[1].x;
	cov3D[4] = T[1].y;
	cov3D[5] = T[1].z;
	cov3D[6] = T[2].x;
	cov3D[7] = T[2].y;
	cov3D[8] = T[2].z;
	normal = {tn.x, tn.y, tn.z};
	return true;
}

__device__ bool computeCenter(const float *cov3D, float2 & center, float2 & extent) {
	glm::mat4x3 T = glm::mat4x3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[3], cov3D[4], cov3D[5],
		cov3D[6], cov3D[7], cov3D[8],
		cov3D[6], cov3D[7], cov3D[8]
	);

	float d = glm::dot(glm::vec3(1.0, 1.0, -1.0), T[3] * T[3]);
	
	if (d == 0.0f) return false;

	glm::vec3 f = glm::vec3(1.0, 1.0, -1.0) * (1.0f / d);

	glm::vec3 p = glm::vec3(
		glm::dot(f, T[0] * T[3]),
		glm::dot(f, T[1] * T[3]), 
		glm::dot(f, T[2] * T[3]));
	
	glm::vec3 h0 = p * p - 
		glm::vec3(
			glm::dot(f, T[0] * T[0]),
			glm::dot(f, T[1] * T[1]), 
			glm::dot(f, T[2] * T[2])
		);

	glm::vec3 h = sqrt(max(glm::vec3(0.0), h0)) + glm::vec3(0.0, 0.0, 1e-2);
	center = {p.x, p.y};
	extent = {h.x, h.y};
	return true;
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const glm::vec2* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	// const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, const float tan_fovy,
	const float focal_x, const float focal_y,
	int* radii,
	float2* points_xy_image,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	glm::vec3 p_world = glm::vec3(orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2]);
	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;
	
	float4 intrins = {focal_x, focal_y, float(W)/2.0, float(H)/2.0};
	glm::vec2 scale = scales[idx];
	glm::vec4 quat = rotations[idx];
	// view frustum cullling TODO
	const float* cov3D;
	bool ok;
	float3 normal;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 9;
	}
	else
	{
		ok = computeCov3D(p_world, quat, scale, viewmatrix, intrins, tan_fovx, tan_fovy, cov3Ds + idx * 9, normal);
		if (!ok) return;
		cov3D = cov3Ds + idx * 9;
	}
	
	//  compute center and extent
	float2 center;
	float2 extent;
	ok = computeCenter(cov3D, center, extent);
	if (!ok) return;

	// add the bounding of countour
#if TIGHTBBOX
	// the effective extent is now depended on the opacity of gaussian.
	float truncated_R = sqrtf(max(9.f + logf(opacities[idx]), 0.000001));
	// if (truncated_R < 1.0) printf("%.2f\n", truncated_R);
#else
	float truncated_R = 3.f;
#endif
	float radius = ceil(truncated_R * max(max(extent.x, extent.y), FilterSize));

	uint2 rect_min, rect_max;
	getRect(center, radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// compute colors 
	// if (colors_precomp == nullptr) {
	glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
	rgb[idx * C + 0] = result.x;
	rgb[idx * C + 1] = result.y;
	rgb[idx * C + 2] = result.z;
	// }

	// assign values
	depths[idx] = p_view.z;
	radii[idx] = (int)radius;
	points_xy_image[idx] = center;
	conic_opacity[idx] = {normal.x, normal.y, normal.z, opacities[idx]};
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);

	// if (idx % 32 == 0) {
	//     printf("%d center %.4f %.4f\n", idx, center.x, center.y);
	//     printf("%d extent %.4f %.4f %.4f\n", idx, extent.x, extent.y);
	// }
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,
	const float* __restrict__ features_misc,
	const float* __restrict__ cov3Ds,
	const float* __restrict__ depths,
	const float4* __restrict__ conic_opacity,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	const int channel_misc,
	float* __restrict__ out_color,
	float* __restrict__ out_depth,
	float* __restrict__ out_misc)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x + 0.5, (float)pix.y + 0.5};

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];
	__shared__ float3 collected_Tu[BLOCK_SIZE];
	__shared__ float3 collected_Tv[BLOCK_SIZE];
	__shared__ float3 collected_Tw[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };


#if RENDER_AXUTILITY
	// render axutility ouput
	float D = { 0 };
	float N[3] = {0};
	float dist1 = {0};
	float dist2 = {0};
	float distortion = {0};
	float max_depth = {0};
	float max_weight = {0};
	float max_contributor = {-1};

#endif

	extern __shared__ float acc_miscs[];
	// float misc[3] = { 0 };
	float* misc = acc_miscs + channel_misc * block.thread_rank();
	for (int ch = 0; ch < channel_misc; ch++)
		misc[ch] = 0;

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
			collected_Tu[block.thread_rank()] = {cov3Ds[9 * coll_id+0], cov3Ds[9 * coll_id+1], cov3Ds[9 * coll_id+2]};
			collected_Tv[block.thread_rank()] = {cov3Ds[9 * coll_id+3], cov3Ds[9 * coll_id+4], cov3Ds[9 * coll_id+5]};
			collected_Tw[block.thread_rank()] = {cov3Ds[9 * coll_id+6], cov3Ds[9 * coll_id+7], cov3Ds[9 * coll_id+8]};
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// compute ray-splat intersection
			// float2 xy = collected_xy[j];
			float3 Tu = collected_Tu[j];
			float3 Tv = collected_Tv[j];
			float3 Tw = collected_Tw[j];
			// compute two planes intersection as the ray intersection
			float3 k = {-Tu.x + pixf.x * Tw.x, -Tu.y + pixf.x * Tw.y, -Tu.z + pixf.x * Tw.z};
			float3 l = {-Tv.x + pixf.y * Tw.x, -Tv.y + pixf.y * Tw.y, -Tv.z + pixf.y * Tw.z};

			float3 p = crossProduct(k, l);

#if SKIL_NEGATIVE
			if (p.z == 0.0) continue; // there is not intersection
#endif

			float2 s = {p.x / p.z, p.y / p.z};
			float rho3d = (s.x * s.x + s.y * s.y); // splat distance
			
			// add low pass filter according to Botsch et al. [2005]. 
			float2 xy = collected_xy[j];
			float2 d = {xy.x - pixf.x, xy.y - pixf.y};
			float rho2d = FilterInvSquare * (d.x * d.x + d.y * d.y); // screen distance
			float rho = min(rho3d, rho2d);
			
			// compute accurate depth when necessary
#if RENDER_AXUTILITY and INTERSECT_DEPTH
			// float depth = (s.x * Tw.x + s.y * Tw.y) + Tw.z;
			float depth = (rho3d <= rho2d) ? (s.x * Tw.x + s.y * Tw.y) + Tw.z : Tw.z; // splat depth
#if SKIL_NEGATIVE
			if (depth < NEAR_PLANE) continue;
#endif
#else
			float depth = Tw.z; // center depth
#endif
			float4 con_o = collected_conic_opacity[j];
			float normal[3] = {con_o.x, con_o.y, con_o.z};
			float power = -0.5f * rho;
			// power = -0.5f * 100.f * max(rho - 1, 0.0f);
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			float weight = alpha * T;

#if RENDER_AXUTILITY
// render distortion map
			float A = 1-T;
#if MAPPED_Z
			float mapped_depth = (FAR_PLANE * depth - FAR_PLANE * NEAR_PLANE) / ((FAR_PLANE - NEAR_PLANE) * depth);
#else		
			float mapped_depth = depth;
#endif
			float error = mapped_depth * mapped_depth * A + dist2 - 2 * mapped_depth * dist1;
			distortion += error * weight;

			// if (alpha * T >= max_weight) {
			if (T > 0.5) {
				max_depth = depth;
				max_weight = weight;
				max_contributor = contributor;
			}

#if DEBUG
			if (collected_id[j] > 0 && pix.x == W / 4 && pix.y == H / 2) {
				printf("%d forward %d %d\n", contributor, pix.x, pix.y);
				printf("%d forward %d normal %.4f %.4f %.4f\n", contributor, normal[0], normal[1], normal[2]);
				printf("%d forward %d A %.8f\n", contributor, collected_id[j], A);
				printf("%d forward %d depth %.8f\n", contributor, collected_id[j], depth);
				printf("%d forward %d D %.8f\n", contributor, collected_id[j], D);
				printf("%d forward %d alpha %.8f\n", contributor, collected_id[j], alpha);
				printf("%d forward %d last_alpha %.8f\n", contributor, collected_id[j], 1-T);
				printf("%d forward %d A %.8f\n", contributor, collected_id[j], A);
				printf("%d forward %d error %.8f\n", contributor, collected_id[j], error);
				printf("%d forward %d loss %.8f\n", contributor, collected_id[j], distortion);
				printf("%d forward %d map_depth %.8f\n", contributor, collected_id[j], mapped_depth);
				printf("%d forward %d contrib %d\n", contributor, collected_id[j], max_contributor);
				printf("-----------\n");
			}
#endif
			// render normal map
			for (int ch=0; ch<3; ch++) N[ch] += normal[ch] * weight;

			// render depth map
			D += depth * weight;
			// mapped depth
			dist1 += mapped_depth * weight;
			dist2 += mapped_depth * mapped_depth * weight;
#endif

			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * weight;
			for (int ch = 0; ch < channel_misc; ch++)
				misc[ch] += features_misc[collected_id[j] * channel_misc + ch] * weight;
			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		for (int ch = 0; ch < channel_misc; ch++)
			out_misc[ch * H * W + pix_id] = misc[ch];

#if RENDER_AXUTILITY
		n_contrib[pix_id + H * W] = max_contributor;
		final_T[pix_id + H * W] = dist1;
		final_T[pix_id + 2 * H * W] = dist2;
		out_depth[pix_id + DEPTH_OFFSET * H * W] = D;
		out_depth[pix_id + ALPHA_OFFSET * H * W] = 1 - T;
		for (int ch=0; ch<3; ch++) out_depth[pix_id + (NORMAL_OFFSET+ch) * H * W] = N[ch];
		out_depth[pix_id + MAXDEPTH_OFFSET * H * W] = max_depth;
		out_depth[pix_id + DISTORTION_OFFSET * H * W] = distortion;
		out_depth[pix_id + MAX_WEIGHT_OFFSET * H * W] = max_weight;
#endif
	}
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float2* means2D,
	const float* colors,
	const float* colors_precomp,
	const float* cov3Ds,
	const float* depths,
	const float4* conic_opacity,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	const int channel_misc,
	float* out_color,
	float* out_depth,
	float* out_misc)
{
	renderCUDA<NUM_CHANNELS> << <grid, block, channel_misc * BLOCK_SIZE * sizeof(float)>> > (
		ranges,
		point_list,
		W, H,
		focal_x, focal_y,
		means2D,
		colors,
		colors_precomp,
		cov3Ds,
		depths,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		channel_misc,
		out_color,
		out_depth,
		out_misc);
}

void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec2* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	// const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, const int H,
	const float focal_x, const float focal_y,
	const float tan_fovx, const float tan_fovy,
	int* radii,
	float2* means2D,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		cov3D_precomp,
		// colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		cov3Ds,
		rgb,
		conic_opacity,
		grid,
		tiles_touched,
		prefiltered
		);
}
